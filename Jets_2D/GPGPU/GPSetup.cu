// File Headers
#include "Jets_2D/GPGPU/GPSetup.hpp"

// Standard Library
#include <iostream>

// CUDA
#include <hip/hip_runtime.h>
#include <>

// Project Headers
#include "Jets_2D/GPGPU/State.hpp"
#include "Jets_2D/GPGPU/GPErrorCheck.hpp"

CraftState* Crafts;

MatchState* Match;
temp* Temp;
config* d_Config;
GraphicsObjectPointer   Buffer;     // Filled by CUDA_Map and copied to global memory
GraphicsObjectPointer* d_Buffer;    // Global memory version

config* h_Config;   // Host side variable. Requirement, whenever this is changed, it must be uploaded to GPU.

bool h_AllDone = false;  // Breaks up epoch iterations so as to not trip Windows GPU watchdog timer and also to allow real-time rendering

namespace Mem
{
    void Setup()
    {
        cudaCheck(hipMalloc(&Match, sizeof(MatchState)));

        cudaCheck(hipMalloc(&Crafts, sizeof(CraftState)));
        cudaCheck(hipDeviceSynchronize());

        // std::cout << "Neuron Address: " << &Crafts->Neuron << "-0x" << std::hex << (unsigned long)(&Crafts->Neuron) + sizeof(float) * 2 * CRAFT_COUNT * NEURON_AMOUNT << ", Weight Address: "  << &Crafts->Weight << "-0x" << std::hex << (unsigned long)(&Crafts->Weight) + sizeof(float) * CRAFT_COUNT * WEIGHT_AMOUNT << std::endl;

        cudaCheck(hipMalloc(&Temp, sizeof(temp)));
        cudaCheck(hipDeviceSynchronize());

        h_Config = new config();

        cudaCheck(hipMalloc(&d_Config, sizeof(config)));
        cudaCheck(hipMemcpy(d_Config, h_Config, sizeof(config), hipMemcpyHostToDevice));
        cudaCheck(hipDeviceSynchronize());

        cudaCheck(hipMalloc(&d_Buffer, sizeof(GraphicsObjectPointer)));
        cudaCheck(hipDeviceSynchronize());
    }
    void Shutdown()
    {
        cudaCheck(hipFree(Match));
        cudaCheck(hipFree(Crafts));
        cudaCheck(hipFree(Temp));

        delete h_Config;
        cudaCheck(hipFree(d_Config));
    }
}
