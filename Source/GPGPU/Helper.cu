#include "Helper.h"

// CUDA
#include <hip/hip_runtime.h>
#include <>

__host__ __device__ int Div_Up(int a, int b)
{
    return (a + b - 1) / b;
}