// File Headers
#include "GPSetup.h"

// CUDA
#include "hip/hip_runtime.h"
#include ""

// Project Headers
#include "GPGPU/State.h"
#include "GPGPU/GPErrorCheck.h"

CraftState* Crafts;

MatchState* Match;
temp* Temp;
config* d_Config;
GraphicsObjectPointer	Buffer;		// Filled by CUDA_Map and copied to global memory
GraphicsObjectPointer* d_Buffer;	// Global memory version

config* h_Config;	// Host side variable. Requirement, whenever this is changed, it must be uploaded to GPU.

bool h_AllDone = false;	 // Breaks up epoch iterations so as to not trip Windows GPU watchdog timer and also to allow real-time rendering

namespace Mem
{
	void Setup()
	{
		cudaCheck(hipMalloc(&Match, sizeof(MatchState)));

		cudaCheck(hipMalloc(&Crafts, sizeof(CraftState)));
		cudaCheck(hipDeviceSynchronize());

		cudaCheck(hipMalloc(&Temp, sizeof(temp)));
		cudaCheck(hipDeviceSynchronize());

		h_Config = new config();

		cudaCheck(hipMalloc(&d_Config, sizeof(config)));
		cudaCheck(hipMemcpy(d_Config, h_Config, sizeof(config), hipMemcpyHostToDevice));
		cudaCheck(hipDeviceSynchronize());

		cudaCheck(hipMalloc(&d_Buffer, sizeof(GraphicsObjectPointer)));
		cudaCheck(hipDeviceSynchronize());
	}
	void Shutdown()
	{
		cudaCheck(hipFree(Match));
		cudaCheck(hipFree(Crafts));
		cudaCheck(hipFree(Temp));

		delete h_Config;
		cudaCheck(hipFree(d_Config));
	}
}
