// Standard Library
#include <iostream>
#include <iomanip>
#include <vector>
#include <cmath>
#include <chrono>
#include <stdint.h>
#include <stdlib.h>
#include <istream>
#include <fstream>
#include <ostream>
#include <sstream>
#include <stdlib.h>

// OpenGL
#include "glad/glad.h"
#include "GLFW/glfw3.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/type_ptr.hpp"

// CUDA
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>
 
// ImGUI
#include "imgui/imgui.h"
#include "imgui/imgui_internal.h"
#include "imgui/imgui_impl_glfw.h"
#include "imgui/imgui_impl_opengl3.h"

// Project Headers
#include "ErrorCheck.h"
#include "GL/GLSetup.h"
#include "GPGPU/Round.h"
#include "GPGPU/GPSetup.h"
#include "GPGPU/MapVertexBuffer.h"
#include "GPGPU/Match.h"
#include "GPGPU/State.h"
#include "GUI/GUI.h"

int main()
{
	// Startup
	Timer = std::chrono::steady_clock::now();
	GL::Setup();
	Mem::Setup();
	Setup();
	Graphics::Setup();
	Init<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
	cudaCheck(hipDeviceSynchronize());

	// Output estimated memory usage for backpropagating advantage function
	unsigned long long StateSize = (sizeof(CraftState) - sizeof(float) * WEIGHT_COUNT - 2 * ((NEURON_COUNT + 1 + 1) * sizeof(float) - sizeof(hiprandState))) *  2 / 1024 / 1024;
	std::cout << "Size of state: " << StateSize << " MB" << std::endl;

	unsigned long long AdvantageSize = StateSize * FRAMERATE_PHYSICS * int(TIME_MATCH);
	std::cout << "GPU memory required: " << AdvantageSize << " MB" << std::endl;

	std::cout << "Number of Layers: " << LAYER_AMOUNT << std::endl;
	std::cout << "Number of Neurons: " << NEURON_COUNT << std::endl;

	TimerStartup = float(std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - Timer).count()) / 1000.f;

	// Game Loop
	while (!glfwWindowShouldClose(window))
	{
		int h_TournamentEpochNumber = 0;
		cudaCheck(hipMemcpy(&Match->TournamentEpochNumber, &h_TournamentEpochNumber, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipDeviceSynchronize());

		// Original Side of the Circle
		for (int i = 0; i < OPPONENT_COUNT && !glfwWindowShouldClose(window); i++)
			Round();

		RoundAssignPlace<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		// TODO: Build 1 kernel from these
		for (int i = 0; i < 10; i++)
		{
			RoundTieFix<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
			cudaCheck(hipDeviceSynchronize());
		}
		
		RoundEnd();

		h_Config->RoundNumber = RoundNumber;
		SyncConfigArray();

		IDAssign<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, d_Config);
		cudaCheck(hipDeviceSynchronize());

		IDTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		IDTransfer<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		int ScoreCumulative[CRAFT_COUNT];
		cudaCheck(hipMemcpy(ScoreCumulative, Crafts->ScoreCumulative, CRAFT_COUNT * sizeof(int), hipMemcpyDeviceToHost));

		int ID[CRAFT_COUNT];
		cudaCheck(hipMemcpy(ID, Crafts->ID, CRAFT_COUNT * sizeof(int), hipMemcpyDeviceToHost));

		int Place[CRAFT_COUNT];
		cudaCheck(hipMemcpy(Place, Crafts->Place, CRAFT_COUNT * sizeof(int), hipMemcpyDeviceToHost));

		float FirstWeight1[CRAFT_COUNT];
		for (int i = 0; i < CRAFT_COUNT; i++)
			cudaCheck(hipMemcpy(&FirstWeight1[i], &Crafts->Weights[i], sizeof(float), hipMemcpyDeviceToHost));

		float ThirtiethWeight1[CRAFT_COUNT];
		for (int i = 0; i < CRAFT_COUNT; i++)
			cudaCheck(hipMemcpy(&ThirtiethWeight1[i], &Crafts->Weights[29 * WEIGHT_COUNT + i], sizeof(float), hipMemcpyDeviceToHost));
		
		WeightsTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, Temp);
		cudaCheck(hipDeviceSynchronize());

		WeightsTransfer<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, Temp);
		cudaCheck(hipDeviceSynchronize());

		WeightsMutate<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, d_Config);
		cudaCheck(hipDeviceSynchronize());

		ScoreTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		ScoreTransfer<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		int ScoreCumulative2[CRAFT_COUNT];
		cudaCheck(hipMemcpy(ScoreCumulative2, Crafts->ScoreCumulative, CRAFT_COUNT * sizeof(int), hipMemcpyDeviceToHost));

		float FirstWeight2[CRAFT_COUNT];
		for (int i = 0; i < CRAFT_COUNT; i++)
			cudaCheck(hipMemcpy(&FirstWeight2[i], &Crafts->Weights[i], sizeof(float), hipMemcpyDeviceToHost));

		float ThirtiethWeight2[CRAFT_COUNT];
		for (int i = 0; i < CRAFT_COUNT; i++)
			cudaCheck(hipMemcpy(&ThirtiethWeight2[i], &Crafts->Weights[29 * WEIGHT_COUNT + i], sizeof(float), hipMemcpyDeviceToHost));

		std::cout << "Results" << std::endl;
		for (int i = 0; i < CRAFT_COUNT; i++)
			std::cout << std::setw(3) << i << " ID: " << std::setw(5) << ID[i] 
			<< " Score: " << std::setw(4) << ScoreCumulative[i] 
			<< " Place: " << std::setw(3) << Place[i] 
			<< " 1st Weight: " << std::setw(12) << FirstWeight1[i]
			<< " 30th Weight: " << std::setw(12) << ThirtiethWeight1[i]
			<< " Sorted:: Score: " << std::setw(5) << ScoreCumulative2[i]
			<< " 1st Weight: " << std::setw(12) << FirstWeight2[i]
			<< " 30th Weight: " << std::setw(12) << ThirtiethWeight2[i] << std::endl;

		/*std::cout << "Score" << std::endl;
		for (int i = 0; i < CRAFT_COUNT; i++)
			std::cout << std::setw(3) << i << " Score: " << std::setw(5) << ScoreCumulative2[i] << std::endl;*/

		ResetScoreCumulative<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		RoundEnd2();
	}

	// Cleanup
	Mem::Shutdown();
	Shutdown();
	Graphics::Shutdown();

	return 0;
}