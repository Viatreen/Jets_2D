// Standard Library
#include <iostream>
#include <iomanip>
#include <vector>
#include <cmath>
#include <chrono>
#include <stdint.h>
#include <stdlib.h>
#include <istream>
#include <fstream>
#include <ostream>
#include <sstream>
#include <stdlib.h>

// OpenGL
#include "glad/glad.h"
#include "GLFW/glfw3.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/type_ptr.hpp"

// CUDA
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>
 
// ImGUI
#include "imgui/imgui.h"
#include "imgui/imgui_internal.h"
#include "imgui/imgui_impl_glfw.h"
#include "imgui/imgui_impl_opengl3.h"

// Project Headers
#include "ErrorCheck.h"
#include "GL/GLSetup.h"
#include "GPGPU/Round.h"
#include "GPGPU/GPSetup.h"
#include "GPGPU/MapVertexBuffer.h"
#include "GPGPU/Match.h"
#include "GPGPU/State.h"
#include "GUI/GUI.h"

int main()
{
	// Startup
	Timer = std::chrono::steady_clock::now();
	GL::Setup();
	Mem::Setup();
	Setup();
	Graphics::Setup();
	Init<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
	cudaCheck(hipDeviceSynchronize());

	// Output estimated memory usage for backpropagating advantage function
	unsigned long long StateSize = (sizeof(CraftState) - sizeof(float) * WARP_SIZE * WEIGHT_COUNT - 2 * WARP_SIZE * ((NEURON_COUNT + 1 + 1) * sizeof(float) - sizeof(hiprandState))) * WARP_COUNT / 2 / 1024 / 1024;
	std::cout << "Size of state: " << StateSize << " MB" << std::endl;

	unsigned long long AdvantageSize = StateSize * FRAMERATE_PHYSICS * int(TIME_MATCH);
	std::cout << "GPU memory required: " << AdvantageSize << " MB" << std::endl;

	std::cout << "Number of Layers: " << LAYER_AMOUNT << std::endl;
	std::cout << "Number of Neurons: " << NEURON_COUNT << std::endl;

	TimerStartup = float(std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - Timer).count()) / 1000.f;

	// Game Loop
	while (!glfwWindowShouldClose(window))
	{
		int h_TournamentEpochNumber = 0;
		cudaCheck(hipMemcpy(&Match->TournamentEpochNumber, &h_TournamentEpochNumber, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipDeviceSynchronize());

		// Original Side of the Circle
		for (int i = 0; i < OPPONENT_COUNT && !glfwWindowShouldClose(window); i++)
			Round();

		RoundAssignPlace<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		// TODO: Build 1 kernel from these
		for (int i = 0; i < 10; i++)
		{
			RoundTieFix<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
			cudaCheck(hipDeviceSynchronize());
		}
		
		RoundEnd();

		h_Config->RoundNumber = RoundNumber;
		SyncConfigArray();

		IDAssign<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, d_Config);
		cudaCheck(hipDeviceSynchronize());

		IDTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		IDTransfer<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		int ScoreCumulative[CRAFT_COUNT];
		for (int i = 0; i < WARP_COUNT; i++)
			cudaCheck(hipMemcpy(&ScoreCumulative[WARP_SIZE * i], CraftsDevicePointers.Warp[i]->ScoreCumulative, WARP_SIZE * sizeof(int), hipMemcpyDeviceToHost));

		int ID[WARP_SIZE];
		cudaCheck(hipMemcpy(ID, CraftsDevicePointers.Warp[0]->ID, WARP_SIZE * sizeof(int), hipMemcpyDeviceToHost));

		int Place[WARP_SIZE];
		cudaCheck(hipMemcpy(Place, CraftsDevicePointers.Warp[0]->Place, WARP_SIZE * sizeof(int), hipMemcpyDeviceToHost));

		/*std::cout << "Results" << std::endl;
		for (int i = 0; i < WARP_SIZE; i++)
			std::cout << std::setw(3) << i << " ID: " << std::setw(5) << ID[i] << " Score: " << std::setw(4) << ScoreCumulative[i] << " Place: " << std::setw(3) << Place[i] << std::endl;*/
		
		WeightsTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, Temp);
		cudaCheck(hipDeviceSynchronize());

		WeightsTransfer<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, Temp);
		cudaCheck(hipDeviceSynchronize());

		WeightsMutate<<<FIT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts, d_Config);
		cudaCheck(hipDeviceSynchronize());

		ScoreTempSave<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		ScoreTransfer<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		for (int i = 0; i < WARP_COUNT; i++)
			cudaCheck(hipMemcpy(&ScoreCumulative[WARP_SIZE * i], CraftsDevicePointers.Warp[i]->ScoreCumulative, WARP_SIZE * sizeof(int), hipMemcpyDeviceToHost));

		/*std::cout << "Score" << std::endl;
		for (int i = 0; i < WARP_SIZE; i++)
			std::cout << std::setw(3) << i << " Score: " << std::setw(5) << ScoreCumulative[i] << std::endl;*/

		ResetScoreCumulative<<<CRAFT_COUNT / BLOCK_SIZE, BLOCK_SIZE>>>(Crafts);
		cudaCheck(hipDeviceSynchronize());

		RoundEnd2();
	}

	// Cleanup
	Mem::Shutdown();
	Shutdown();
	Graphics::Shutdown();

	return 0;
}